
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>

using namespace std;


__host__ double* createMatrix(int dim, int zero){
    double *matrix = (double*) malloc(dim * dim * (sizeof(double)));
    
    for (int i = 0; i < dim*dim; i++)
        *(matrix + i) = (zero)? 0.0 : (10.0*rand()/(RAND_MAX+1.0));
    
    return matrix;
}


__host__ double* createMatrixWithZeroes(int dim){
    return createMatrix(dim, 1);
}


__host__ double* createRandomMatrix(int dim){
    return createMatrix(dim, 0);
}


__host__ void printMatrix(double *matrix, int dim){
    
    for (int i = 0; i < dim*dim; i++){
        printf("%.2f ", *(matrix + i));
        
        if (((i + 1) % dim) == 0 )
            printf("\n");
    }
    printf("\n");
}


__device__ double calculateCell(int i, int dim, double *matrixA, double *matrixB){
    double cell = 0.0;
    
    for (int n = 0; n < dim; n++){
        int row = dim*(i/dim) + n;
        int col = dim*(i%dim) + n;
        cell += *(matrixA + row) * *(matrixB + col);
    }
    
    return cell;
}


__global__ void multiplyMatrix(double *matrixA, double *matrixB, double *matrixC,
                               int dim, int limit){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < limit){
        *(matrixC + i) = calculateCell(i, dim, matrixA, matrixB);
    }
}


int main(){
    int dim =  32;
    printf("\n---------------------------------------------\n");
    
    while (dim <= 2048){
        int limit = dim*dim;
        int size = dim * dim * sizeof(double);
        int blocks = 128;
        int threadsPerBlock = dim*dim/blocks;
        
        double *h_matrixA = createRandomMatrix(dim);
        double *h_matrixB = createRandomMatrix(dim);
        double *h_matrixC = createMatrixWithZeroes(dim);
        double *d_matrixA, *d_matrixB, *d_matrixC;
        
        hipMalloc(&d_matrixA, size);
        hipMalloc(&d_matrixB, size);
        hipMalloc(&d_matrixC, size);
        
        hipMemcpy(d_matrixA, h_matrixA, size, hipMemcpyHostToDevice);
        hipMemcpy(d_matrixB, h_matrixB, size, hipMemcpyHostToDevice);
        hipMemcpy(d_matrixC, h_matrixC, size, hipMemcpyHostToDevice);
        
        struct timeval start_time;
        struct timeval end_time;
        gettimeofday(&start_time, NULL);
        multiplyMatrix<<<blocks, threadsPerBlock>>>(d_matrixA, d_matrixB, d_matrixC,
                                                    dim, limit);
        gettimeofday(&end_time, NULL);
        double seconds = (((1000.0*end_time.tv_sec) + (end_time.tv_usec/1000.0)) -
                         ((1000.0*start_time.tv_sec) + (start_time.tv_usec/1000.0)))/1000.0;
        hipMemcpy(h_matrixC, d_matrixC, size, hipMemcpyDeviceToHost);
        
        //printf("\n");
        //printf("matrix a\n");
        //printMatrix(h_matrixA, dim);
        //printf("matrix b\n");
        //printMatrix(h_matrixB, dim);
        //printf("matrix c\n");
        //printMatrix(h_matrixC, dim);
        printf("Taken time for a matrix of %dX%d with %d blocks and %d threads per block: %.5fs\n",
               dim, dim, blocks, threadsPerBlock, seconds);
        
        hipFree(d_matrixA);
        hipFree(d_matrixB);
        hipFree(d_matrixC);
        
        free(h_matrixA);
        free(h_matrixB);
        free(h_matrixC);
        
        printf("---------------------------------------------\n\n");
        printf("---------------------------------------------\n");
        
        dim *= 2;
    }
    
	return 0;
}